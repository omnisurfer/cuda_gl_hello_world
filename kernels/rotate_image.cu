#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t rotateImage(
	const unsigned char* input_image_data,
	unsigned char* output_image_data,
	int x_dimension,
	int y_dimension,
	int stride,
	float angle_degrees	
);

__global__ void rotateImageKernel(
	unsigned char* input_image_data, 
	unsigned char* output_image_data,
	int x_dimension,
	int y_dimension,
	int stride,
	size_t pitch,
	float angle_radians
) {
	//noop
}

extern "C" {

	int execute_image_rotation_kernel(
		const unsigned char* input_image_data, 
		unsigned char* output_image_data,
		int x_dimension, 
		int y_dimension, 
		int stride, 
		float angle_degrees		
	) {

		hipError_t cuda_status = rotateImage(
			input_image_data,
			output_image_data,
			x_dimension,
			y_dimension,
			stride,
			angle_degrees	
		);

		if (cuda_status != hipSuccess) {
			fprintf(stderr, "rotateImage failed to execute!");
			return 1;
		}

		// cudaDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cuda_status = hipDeviceReset();
		if (cuda_status != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
}

hipError_t rotateImage(
	const unsigned char* input_image_data,
	unsigned char* output_image_data,
	int x_dimension,
	int y_dimension,
	int stride,
	float angle_degrees
) {
	
	unsigned char* device_input_buffer = 0;
	unsigned char* device_output_buffer = 0;
	hipError_t cuda_status;

	float angle_radians = angle_degrees * (3.14 / 180.0);

	// choose GPU
	cuda_status = hipSetDevice(0);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed! Selected GPU may not be CUDA compatible.");
	}

	
	size_t device_pitch;

	/* 
	- allocate pitched gpu buffers for one input image and an output image
	- reference https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html?highlight=cudaMallocPitch#device-memory		
	*/ 
	cuda_status = hipMallocPitch(
		&device_input_buffer,
		&device_pitch,
		x_dimension * sizeof(unsigned char) * stride,
		y_dimension
	);

	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMallocPitch failed!");
		goto Error;
	}

	cuda_status = hipMallocPitch(
		&device_output_buffer,
		&device_pitch,
		x_dimension * sizeof(unsigned char) * stride,
		y_dimension
	);

	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMallocPitch failed!");
		goto Error;
	}

	/* 
	- copy input image data from host memory to GPU buffer	
	- for reference https://stackoverflow.com/questions/16119943/how-and-when-should-i-use-pitched-pointer-with-the-cuda-api
	*/
	cuda_status = hipMemcpy2D(
		device_input_buffer,
		device_pitch,
		input_image_data,
		x_dimension * sizeof(unsigned char) * stride,
		x_dimension,
		y_dimension,
		hipMemcpyHostToDevice
	);

	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D failed!");
		goto Error;
	}
		
	// launch kernel
	rotateImageKernel<<<1, device_pitch>>>(
		device_input_buffer, 
		device_output_buffer,
		x_dimension, 
		y_dimension, 
		stride, 
		device_pitch, 
		angle_radians
		);

	// check for errors launching the kernel
	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cuda_status));
		goto Error;
	}

Error:
	hipFree(device_input_buffer);
	hipFree(device_output_buffer);

	return cuda_status;
}