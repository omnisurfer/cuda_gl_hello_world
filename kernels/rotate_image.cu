#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t rotateImage(
	const unsigned char* input_image_data,
	unsigned char* output_image_data,
	int x_dimension,
	int y_dimension,
	int stride,
	float angle_degrees	
);

__global__ void rotateImageKernel(
	unsigned char* input_image_data, 
	unsigned char* output_image_data,
	int x_dimension,
	int y_dimension,
	int stride,
	size_t pitch,
	float angle_radians
) {
	for (int row_index = 0; row_index < y_dimension; ++row_index) {
		unsigned char* input_row_ptr = input_image_data + row_index * pitch;
		unsigned char* output_row_ptr = output_image_data + row_index * pitch;
		for (int column_index = 0; column_index < x_dimension; ++column_index) {
			output_row_ptr[column_index] = input_row_ptr[column_index];
		}
	}	
}

extern "C" {

	int execute_image_rotation_kernel(
		const unsigned char* input_image_data, 
		unsigned char* output_image_data,
		int x_dimension, 
		int y_dimension, 
		int stride, 
		float angle_degrees		
	) {

		hipError_t cuda_status = rotateImage(
			input_image_data,
			output_image_data,
			x_dimension,
			y_dimension,
			stride,
			angle_degrees	
		);

		if (cuda_status != hipSuccess) {
			fprintf(stderr, "rotateImage failed to execute!");
			return 1;
		}

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cuda_status = hipDeviceReset();
		if (cuda_status != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
}

hipError_t rotateImage(
	const unsigned char* input_image_data,
	unsigned char* output_image_data,
	int x_dimension,
	int y_dimension,
	int stride,
	float angle_degrees
) {
	
	unsigned char* device_input_buffer = 0;
	unsigned char* device_output_buffer = 0;
	hipError_t cuda_status;

	float angle_radians = angle_degrees * (3.14 / 180.0);

	// choose GPU
	cuda_status = hipSetDevice(0);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed! Selected GPU may not be CUDA compatible.");
	}

	
	size_t device_pitch;

	/* 
	- allocate pitched gpu buffers for one input image and an output image
	- reference https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html?highlight=hipMallocPitch#device-memory		
	*/ 
	cuda_status = hipMallocPitch(
		&device_input_buffer,
		&device_pitch,
		x_dimension * sizeof(unsigned char) * stride,
		y_dimension
	);

	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMallocPitch failed!");
		goto Error;
	}

	cuda_status = hipMallocPitch(
		&device_output_buffer,
		&device_pitch,
		x_dimension * sizeof(unsigned char) * stride,
		y_dimension
	);

	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMallocPitch failed!");
		goto Error;
	}

	/* 
	- copy input image data from host memory to GPU buffer	
	- for reference
		- https://stackoverflow.com/questions/16119943/how-and-when-should-i-use-pitched-pointer-with-the-cuda-api
		- https://stackoverflow.com/questions/16491232/how-do-i-use-cudamemcpy2d-devicetohost

	*/
	size_t host_pitch = x_dimension * sizeof(unsigned char); // *stride;
	size_t x_dimension_in_bytes = host_pitch;

	cuda_status = hipMemcpy2D(
		device_input_buffer,
		device_pitch,
		input_image_data,
		host_pitch,
		x_dimension_in_bytes,
		y_dimension,
		hipMemcpyHostToDevice
	);

	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D failed: host to device!");
		goto Error;
	}
		
	// launch kernel
	rotateImageKernel<<<1, device_pitch>>>(
		device_input_buffer, 
		device_output_buffer,
		x_dimension, 
		y_dimension, 
		stride, 
		device_pitch, 
		angle_radians
		);

	// check for errors launching the kernel
	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cuda_status));
		goto Error;
	}

	// copy memory back to local buffer
	cuda_status = hipMemcpy2D(
		output_image_data,
		host_pitch,
		device_output_buffer,
		device_pitch,
		x_dimension_in_bytes,
		y_dimension,
		hipMemcpyDeviceToHost
	);

	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D failed: host to device!");
		goto Error;
	}

Error:
	hipFree(device_input_buffer);
	hipFree(device_output_buffer);

	return cuda_status;
}