#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t rotateImage(
	const float* input_image_data,
	float* output_image_data,
	int x_dimension,
	int y_dimension,
	int stride,
	float angle_degrees	
);

__global__ void rotateImageKernel(
	float* input_image_data, 
	float* output_image_data,
	int x_dimension,
	int y_dimension,
	int stride,
	size_t pitch,
	float angle_radians
) {

	/*
	- reference: https://stackoverflow.com/questions/9833316/cuda-image-rotation
	*/

	if (false) {
		printf("block idx/y %u %u, thread idx/y %u %u, blockDim x/y %u %u\n",
			blockIdx.x,
			blockIdx.y,
			threadIdx.x,
			threadIdx.y,
			blockDim.x,
			blockDim.y
		);
	}

	int index_i = blockIdx.x * pitch + threadIdx.x;
	int index_j = blockIdx.y * pitch + threadIdx.y;

	int x_center = x_dimension - x_dimension / 2;
	int y_center = y_dimension - y_dimension / 2;

	// printf("%u, %u\n", index_i, index_j);

	for (int row_index = 0; row_index < y_dimension; ++row_index) {
				
		float* input_row_ptr = (float*)((char*)input_image_data + row_index * pitch);
		float* output_row_ptr = (float*)((char*)output_image_data + row_index * pitch);
		
		for (int column_index = 0; column_index < x_dimension; ++column_index) {

			float modified_value = input_row_ptr[column_index] + 0.5;

			output_row_ptr[column_index] = modified_value; // input_row_ptr[column_index];
			
			if (false) {
				printf("input %f\n", input_row_ptr[column_index]);
			}

			if (false) {
				printf("post idx.x %i idx.y %i out %f in %f\n", 
					index_i, 
					index_j,					
					input_row_ptr[column_index], 
					output_row_ptr[column_index]
				);
			}
		}
	}	
}

extern "C" {

	int execute_image_rotation_kernel(
		const float* input_image_data, 
		float* output_image_data,
		int x_dimension, 
		int y_dimension, 
		int stride, 
		float angle_degrees		
	) {

		hipError_t cuda_status = rotateImage(
			input_image_data,
			output_image_data,
			x_dimension,
			y_dimension,
			stride,
			angle_degrees	
		);

		if (cuda_status != hipSuccess) {
			fprintf(stderr, "rotateImage failed to execute!\n");
			return 1;
		}

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cuda_status = hipDeviceReset();
		if (cuda_status != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!\n");
			return 1;
		}

		return 0;
	}
}

hipError_t rotateImage(
	const float* input_image_data,
	float* output_image_data,
	int x_dimension,
	int y_dimension,
	int stride,
	float angle_degrees
) {
	
	float* device_input_buffer = 0;
	float* device_output_buffer = 0;
	hipError_t cuda_status;

	float angle_radians = angle_degrees * (3.14 / 180.0);

	// choose GPU
	cuda_status = hipSetDevice(0);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed! Selected GPU may not be CUDA compatible.\n");
	}

	
	size_t device_pitch;

	/* 
	- allocate pitched gpu buffers for one input image and an output image
	- reference https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html?highlight=hipMallocPitch#device-memory		
	*/ 
	cuda_status = hipMallocPitch(
		&device_input_buffer,
		&device_pitch,
		x_dimension * sizeof(float),
		y_dimension
	);

	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMallocPitch failed!\n");
		goto Error;
	}

	cuda_status = hipMallocPitch(
		&device_output_buffer,
		&device_pitch,
		x_dimension * sizeof(float),
		y_dimension
	);

	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMallocPitch failed!\n");
		goto Error;
	}

	/* 
	- copy input image data from host memory to GPU buffer	
	- for reference
		- https://stackoverflow.com/questions/16119943/how-and-when-should-i-use-pitched-pointer-with-the-cuda-api
		- https://stackoverflow.com/questions/16491232/how-do-i-use-cudamemcpy2d-devicetohost

	*/
	size_t host_pitch = x_dimension * sizeof(float);
	size_t x_dimension_in_bytes = host_pitch;

	cuda_status = hipMemcpy2D(
		device_input_buffer,
		device_pitch,
		input_image_data,
		host_pitch,
		x_dimension_in_bytes,
		y_dimension,
		hipMemcpyHostToDevice
	);

	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D failed, host to device: %s\n", hipGetErrorString(cuda_status));
		goto Error;
	}
		
	// launch kernel
	rotateImageKernel<<<1, device_pitch>>>(
		device_input_buffer, 
		device_output_buffer,
		x_dimension, 
		y_dimension, 
		stride, 
		device_pitch, 
		angle_radians
		);

	// check for errors launching the kernel
	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cuda_status));
		goto Error;
	}

	// copy memory back to local buffer
	cuda_status = hipMemcpy2D(
		output_image_data,
		host_pitch,
		device_output_buffer,
		device_pitch,
		x_dimension_in_bytes,
		y_dimension,
		hipMemcpyDeviceToHost
	);

	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D failed: host to device!\n");
		goto Error;
	}

Error:
	hipFree(device_input_buffer);
	hipFree(device_output_buffer);

	return cuda_status;
}