#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>

#include <hip/hip_math_constants.h>

hipError_t rotateImage(
	const float* input_image_data,
	float* output_image_data,
	int x_dimension,
	int y_dimension,
	int stride,
	float angle_degrees	
);

__global__ void rotateImageKernel(
	float* input_image_data, 
	float* output_image_data,
	int x_dimension,
	int y_dimension,
	int stride,
	size_t pitch,
	float angle_radians
) {

	/*
	- reference: https://stackoverflow.com/questions/9833316/cuda-image-rotation
	*/

	if (false) {
		printf("block idx/y %u %u, thread idx/y %u %u, blockDim x/y %u %u\n",
			blockIdx.x,
			blockIdx.y,
			threadIdx.x,
			threadIdx.y,
			blockDim.x,
			blockDim.y
		);
	}
	
	int index_i = blockIdx.x * pitch + threadIdx.x;
	int index_j = blockIdx.y * pitch + threadIdx.y;

	int x_center = x_dimension - x_dimension / 2;
	int y_center = y_dimension - y_dimension / 2;
		
	for (int row_index = 0; row_index < y_dimension; ++row_index) {
								
		for (int column_index = 0; column_index < x_dimension; ++column_index) {

			/* unmodified row pointers output_row_ptr indexes unmodified */
			float* input_row_ptr = (float*)((char*)input_image_data + row_index * pitch);
			float* output_row_ptr = (float*)((char*)output_image_data + row_index * pitch);
			
			float raw_new_column_index = ((float)column_index - x_center) * cos(angle_radians) - ((float)row_index - y_center) * sin(angle_radians) + x_center;
			
			if (raw_new_column_index < 1) {
				raw_new_column_index = 0;
			}
			
			int new_column_index = int(round(raw_new_column_index));

			// clamp new_column_index
			if (new_column_index < 1)
			{
				new_column_index = 0;
			}
			else if (new_column_index > x_dimension - 1)
			{
				new_column_index = x_dimension - 1;
			}

			float raw_new_row_index = ((float)column_index - x_center) * sin(angle_radians) + ((float)row_index - y_center) * cos(angle_radians) + y_center;
			
			if (raw_new_row_index < 1)
			{
				raw_new_row_index = 0;
			}

			int new_row_index = int(round(raw_new_row_index));

			// clamp new_row_index
			if (new_row_index < 1)
			{
				new_row_index = 0;
			}
			else if (new_row_index > y_dimension - 1)
			{
				new_row_index = y_dimension - 1;
			}

			/* input_row_ptr must point to where I want to read a pixel from after transform */
			float* new_input_row_ptr = (float*)((char*)input_image_data + new_row_index * pitch);

			if (true) {
				
				if (new_column_index >= 0 && new_column_index < y_dimension) {
					
					/**/
					printf("orig x,y %i %i\tnew x,y %i %i\tnew raw %f %f\n", 
						row_index, column_index, new_row_index, new_column_index, raw_new_row_index, raw_new_column_index);
					/**/

					output_row_ptr[column_index] = new_input_row_ptr[new_column_index];
				}
			}			

			if (false) {
				printf("post idx.x %i idx.y %i out %f in %f\n", 
					index_i, 
					index_j,					
					input_row_ptr[column_index], 
					output_row_ptr[column_index]
				);
			}
		}
	}	
}

extern "C" {

	int execute_image_rotation_kernel(
		const float* input_image_data, 
		float* output_image_data,
		int x_dimension, 
		int y_dimension, 
		int stride, 
		float angle_degrees		
	) {

		hipError_t cuda_status = rotateImage(
			input_image_data,
			output_image_data,
			x_dimension,
			y_dimension,
			stride,
			angle_degrees	
		);

		if (cuda_status != hipSuccess) {
			fprintf(stderr, "rotateImage failed to execute!\n");
			return 1;
		}

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cuda_status = hipDeviceReset();
		if (cuda_status != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!\n");
			return 1;
		}

		return 0;
	}
}

hipError_t rotateImage(
	const float* input_image_data,
	float* output_image_data,
	int x_dimension,
	int y_dimension,
	int stride,
	float angle_degrees
) {
	
	float* device_input_buffer = 0;
	float* device_output_buffer = 0;
	hipError_t cuda_status;

	float angle_radians = angle_degrees * (HIP_PI_F / 180.0);

	// choose GPU
	cuda_status = hipSetDevice(0);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed! Selected GPU may not be CUDA compatible.\n");
	}

	
	size_t device_pitch;

	/* 
	- allocate pitched gpu buffers for one input image and an output image
	- reference https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html?highlight=hipMallocPitch#device-memory		
	*/ 
	cuda_status = hipMallocPitch(
		&device_input_buffer,
		&device_pitch,
		x_dimension * sizeof(float),
		y_dimension
	);

	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMallocPitch failed!\n");
		goto Error;
	}

	cuda_status = hipMallocPitch(
		&device_output_buffer,
		&device_pitch,
		x_dimension * sizeof(float),
		y_dimension
	);

	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMallocPitch failed!\n");
		goto Error;
	}

	/* 
	- copy input image data from host memory to GPU buffer	
	- for reference
		- https://stackoverflow.com/questions/16119943/how-and-when-should-i-use-pitched-pointer-with-the-cuda-api
		- https://stackoverflow.com/questions/16491232/how-do-i-use-cudamemcpy2d-devicetohost

	*/
	size_t host_pitch = x_dimension * sizeof(float);
	size_t x_dimension_in_bytes = host_pitch;

	cuda_status = hipMemcpy2D(
		device_input_buffer,
		device_pitch,
		input_image_data,
		host_pitch,
		x_dimension_in_bytes,
		y_dimension,
		hipMemcpyHostToDevice
	);

	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D failed, host to device: %s\n", hipGetErrorString(cuda_status));
		goto Error;
	}
	
	int compute_pitch = 1; // device_pitch;

	// launch kernel
	rotateImageKernel<<<1, compute_pitch>>>(
		device_input_buffer, 
		device_output_buffer,
		x_dimension, 
		y_dimension, 
		stride, 
		device_pitch, 
		angle_radians
		);

	// check for errors launching the kernel
	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cuda_status));
		goto Error;
	}

	// copy memory back to local buffer
	cuda_status = hipMemcpy2D(
		output_image_data,
		host_pitch,
		device_output_buffer,
		device_pitch,
		x_dimension_in_bytes,
		y_dimension,
		hipMemcpyDeviceToHost
	);

	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D failed: host to device!\n");
		goto Error;
	}

Error:
	hipFree(device_input_buffer);
	hipFree(device_output_buffer);

	return cuda_status;
}